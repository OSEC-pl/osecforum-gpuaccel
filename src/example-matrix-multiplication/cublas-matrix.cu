#include <stdio.h>
#include <stdint.h>

#include <cuda/hip/hip_runtime.h>
#include <cuda/hipblas.h>

#define MATRIX_EDGE_SIZE 5000 

hipblasHandle_t handle;

/*
 * Allocate memory for m * n sized matrix.
 */
float *
matrix_alloc(uint32_t m, uint32_t n)
{
	float *matrix;

	hipMallocManaged(&matrix, m * n * sizeof(hipComplex));

	return matrix;
}

/*
 * Fill the matrix with numbers. Not random at all.
 */
void
matrix_fill(float *matrix, uint32_t m, uint32_t n)
{
	matrix[0] = 10;
	matrix[1] = 10;
}

/*
 * Multiply matrices.
 */
float *
matrix_multiply(float *a, float *b, uint32_t m, uint32_t n)
{
	float *c;

	float alpha, beta;
       
	alpha = 1.0f;
	beta = 1.0f;	

	c = matrix_alloc(m, n);

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
	    MATRIX_EDGE_SIZE, MATRIX_EDGE_SIZE, MATRIX_EDGE_SIZE, &alpha, a, MATRIX_EDGE_SIZE, b, MATRIX_EDGE_SIZE, &beta, c, MATRIX_EDGE_SIZE);

	hipDeviceSynchronize ();

	return c;
}

int
main()
{
	float *a, *b, *c;
	uint32_t i, j, k;


	a = matrix_alloc(MATRIX_EDGE_SIZE, MATRIX_EDGE_SIZE);
	b = matrix_alloc(MATRIX_EDGE_SIZE, MATRIX_EDGE_SIZE);
	matrix_fill(a, MATRIX_EDGE_SIZE, MATRIX_EDGE_SIZE);
	matrix_fill(b, MATRIX_EDGE_SIZE, MATRIX_EDGE_SIZE);

	hipblasCreate (&handle);

	c = matrix_multiply(a, b, MATRIX_EDGE_SIZE, MATRIX_EDGE_SIZE);

	i = 0; k = 0;
	for (j = 0; j < 5; j++) {
		k = j * MATRIX_EDGE_SIZE;
		for (i = 0; i < 5; i++) {
			printf("%.2f\t", c[k]);
			k++;
		}
		printf("\n");
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}

